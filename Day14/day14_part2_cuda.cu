#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <assert.h>
#include <ctype.h>
#include <stddef.h>
#include <unistd.h>
#include <math.h>
#include <hip/hip_runtime.h>


#ifndef N
#define N (100)
#endif

__device__ int superior(char a, char b)
{
  return (a > b);
}

__device__ int inferior(char a, char b)
{
  return (a < b);
}

__device__ void my_compare_and_swap(char curr[N], char *next,  int (*op)(char, char))
{
#ifdef DEBUG
  assert(curr);
  assert(next);
#endif		
  if (*next != '#'){
    if (op(*curr, *next)){ 
      char temp = *curr;
      *curr = *next;
      *next = temp;
    }
  }
}
__device__ void my_sort_north(char *base, int size)
{
  if (size > 1) {
    for (int i = 0; i < size -1 ; i++){
      my_compare_and_swap(base+i,base+(i+1),superior);
    }
    my_sort_north(base,size-1);
  }
}

__device__ void my_sort_south(char *base, int size, int min)
{  
  if (min < size - 1) {
    for (int i = size - 1 ; i > min ; i--){
      my_compare_and_swap(base+(i),base+(i-1),superior);
    }
    my_sort_south(base,size,min+1);
  }
}

__device__ void my_sort_east(char *base, int coord, int size, int stride)
{
  if (size > 1) {
    for (int i = 0; i < size -1 ; i++){
      my_compare_and_swap(base+i*stride+coord,base+(i+1)*stride+coord,superior);
    }
    my_sort_east(base,coord,size-1,stride);
  }
}

__device__ void my_sort_west(char *base, int coord, int size, int stride, int min)
{  
  if (min < size - 1) {
    for (int i = size - 1 ; i > min ; i--){
      my_compare_and_swap(base+i*stride+coord,base+(i-1)*stride+coord,superior);
    }
    my_sort_west(base,coord,size,stride,min+1);
  }
}
  
void print_bar(char c, int size, FILE *file, int flush)
{
  char str[size+1];
  memset(str,0,(size+1)*sizeof(char));
  for(int i = 0 ; i < size ;i++){
    sprintf(str+i,"%c",c);
  }
  
  fprintf(file,"%s%s",str,(flush) ? "\n" : "");
}

__global__ void sortN(char *base, int size) {
  int tid = threadIdx.x;  
  
  if (tid < size){    
   my_sort_north((base+tid*size), size);
  }
}

__global__ void sortW(char *base, int size) {
  int tid = threadIdx.x;    
  if (tid < size){
    my_sort_west(base,tid,size,size,0);
  }
}

__global__ void sortS(char *base, int size) {
  int tid = threadIdx.x;    
   if (tid < size){
    my_sort_south((base+tid*size), size, 0);
   }
}

__global__ void sortE(char *base, int size) {
  int tid = threadIdx.x;    
  if (tid < size){
    my_sort_east(base, tid, size, size);
  }
}

int main(int argc, char *argv[])
{
  if (argc < 3){
    fprintf(stdout,"usage: ./day14_part2 <input_file> <num_cycles>\n");
    exit(EXIT_FAILURE);
  } else {    
    char *buffer = NULL;
    size_t size = 0;
    ssize_t ret = 0;
    
    FILE *file = fopen(argv[1],"r");
    assert(file);

    long int load = 0;

    int x_max = 0;
    int y_max = 0;
    
    while ((ret = getline(&buffer,&size,file)) != -1) {      
      buffer[strlen(buffer)-1] = '\0'; //remove EOL character
      x_max = strlen(buffer);
      char *temp = (char *)calloc((N+1),sizeof(char)); 
      memcpy(temp,buffer,N+1);    	
      fprintf(stdout,"(Line %3i): %s\n",N,temp);      
      free(temp);
      y_max++;
    }
    rewind(file);
    fprintf(stdout,"========== Matrix is (%3i)x(%3i)\n",N,N);


    char data[N][N];
    memset(data,0,sizeof(char)*(N)*(N)); // \0 everywhere

    char copy[N][N]; 
    memset(copy,0,sizeof(char)*(N)*(N)); // \0 everywhere
    
    //rotate matrix so that columns can be handled
    //directly as arrays
    for(int y = (N-1) ; y >= 0  ; y--){
      ret = getline(&buffer,&size,file);
      if( ret != -1){	
	for(int x = 0 ; x < N ; x++){
	  data[x][y] = buffer[x];
	}	
      }
    }

    for(int y = N-1 ; y >= 0 ; y--){
      fprintf(stdout,"Data  %3i : ",y);
      for(int x = 0 ; x < N ; x++){
	fprintf(stdout,"%c", data[x][y]);
      }
      fprintf(stdout,"\n");
    }

    for(int y = 0 ; y < N ; y++)
      for(int x = 0 ; x < N ; x++)
	copy[x][y] = data[x][y];

    char *gpu_data, *gpu_copy;
    
    // Allocate device memory 
    hipMalloc((void**)&gpu_data, N * N);
    hipMalloc((void**)&gpu_copy, N * N);

    // Transfer data from host to device memory
    hipMemcpy(gpu_data, data, sizeof(char) * N * N, hipMemcpyHostToDevice);
    hipMemcpy(gpu_copy, copy, sizeof(char) * N * N, hipMemcpyHostToDevice);


    //Cycle : N, W, S, E

    long int max = atoi(argv[2]); //1000000000;

    for(long int num_cycles = 0 ; num_cycles < max ; num_cycles++){

      if((num_cycles > 100) &&(num_cycles%(max/100) == 0))
	{
	  fprintf(stdout,".");
	  fflush(stdout);
	}

      sortN <<<1,N>>> (gpu_data, N);
      sortW <<<1,N>>> (gpu_data, N);	
      sortS <<<1,N>>> (gpu_data, N);	
      sortE <<<1,N>>> (gpu_data, N);	
    }

    hipMemcpy(data, gpu_data, sizeof(char) * N * N, hipMemcpyDeviceToHost);
    
    print_bar('=',10,stdout,1);
    for(int y = N -1 ; y >= 0 ; y--){
      fprintf(stdout,"Final  %3i : ",y);
      for(int x = 0 ; x < N ; x++){
	fprintf(stdout,"%c", data[x][y]);
      }
      fprintf(stdout,"\n");
    }
    print_bar('=',10,stdout,1);
    
    //compute load
    for(int y = 0 ; y < N ; y++){
      for(int i = 0 ; i < N ; i++){
	if( (data[i][y]) == 'O'){
	  load += (y+1);
	}	
      }
    }
    
    fprintf(stdout,"========== Load = %li\n",load);


    free(buffer);
    fclose(file);    
  }
  
  exit(EXIT_SUCCESS);
}

  
